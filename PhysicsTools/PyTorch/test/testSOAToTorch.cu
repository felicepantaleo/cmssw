#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cuda/std/array>
#include <cuda/std/ranges>
#include <iostream>
#include <chrono>
#include <array>

#include <hip/hip_runtime.h>
#include <torch/torch.h>
#include <torch/script.h>

#include "testBase.h"

class testSOAToTorch : public testBasePyTorch {
  CPPUNIT_TEST_SUITE(testSOAToTorch);
  CPPUNIT_TEST(test);
  CPPUNIT_TEST_SUITE_END();

public:
  std::string pyScript() const override;
  void test() override;
};

CPPUNIT_TEST_SUITE_REGISTRATION(testSOAToTorch);

std::string testSOAToTorch::pyScript() const { return "create_linear_dnn.py"; }

template <typename T, std::size_t N>
torch::Tensor array_to_tensor(torch::Device device, T* arr, const long int* size) {
  long int arr_size[N];
  long int arr_stride[N];
  std::copy(size, size+N, arr_size);
  std::copy(size, size+N, arr_stride);

  std::shift_right(std::begin(arr_stride), std::end(arr_stride), 1);
  arr_stride[0] = 1;
  arr_stride[N-1] *= arr_stride[N-2];

  auto options = torch::TensorOptions().dtype(torch::CppTypeToScalarType<T>()).device(device).pinned_memory(true);
  torch::Tensor tensor = torch::from_blob(arr, arr_size, arr_stride, options);

  return tensor;
}

template <typename T, std::size_t N>
void print_column_major(T* arr, const long int* size) {
  if (N == 2) {
    for (int i = 0; i < size[0]; i++) {
      for (int j = 0; j < size[1]; j++) {
          std::cout << arr[i + j*size[0]] << " ";
      }
      std::cout << std::endl;
    } 
  } else if (N == 3) {
    for (int i = 0; i < size[0]; i++) {
      std::cout << "(" << i << ", .., ..)" << std::endl;
      for (int j = 0; j < size[1]; j++) {
        for (int k = 0; k < size[2]; k++) {
          std::cout << arr[i + j*size[0] + k*size[0]*size[1]] << " ";
        }
        std::cout << std::endl;
      } 
      std::cout << std::endl;
    }
  }
  std::cout << std::endl;
}


template <typename T, std::size_t N, std::size_t M>
void run(torch::Device device, torch::jit::script::Module model, T* input, const long int* input_shape, T* output, const long int* output_shape) {
  torch::Tensor input_tensor = array_to_tensor<T, N>(device, input, input_shape);

  // from_blod doesn't work if use array from parameter list
  long int res_shape[M];
  std::copy(output_shape, output_shape+M, res_shape);

  std::vector<torch::jit::IValue> inputs{input_tensor};
  auto options = torch::TensorOptions().dtype(torch::CppTypeToScalarType<T>()).device(device).pinned_memory(true);
  // RESULT IS IN ROW MAJOR
  torch::from_blob(output, res_shape, options) = model.forward(inputs).toTensor();
}


void testSOAToTorch::test() {
  torch::Device device(torch::kCUDA);
  
  float input_cpu[] = {1, 2, 3, 2, 2, 4, 4, 3, 1, 3, 1, 2};
  const long int shape[] = {4, 3};

  float result_cpu[4][2];
  float result_check[4][2] = {{2.3, -0.5}, {6.6, 3.0}, {2.5, -4.9}, {4.4, 1.3}};
  const long int result_shape[] = {4, 2};

  // Prints array in correct form.
  print_column_major<float, 2>(input_cpu, shape);

  float *input_gpu, *result_gpu;
  hipMalloc(&input_gpu, sizeof(input_cpu));
  hipMalloc(&result_gpu, sizeof(result_cpu));
  hipMemcpy(input_gpu, input_cpu, sizeof(input_cpu), hipMemcpyHostToDevice); 

  torch::jit::script::Module model;
  try {
    // Deserialize the ScriptModule from a file using torch::jit::load().
    std::string model_path = dataPath_ + "/linear_dnn.pt";
    model = torch::jit::load(model_path);
    model.to(device);


  } catch (const c10::Error& e) {
    std::cerr << "error loading the model\n" << e.what() << std::endl;
  }
  
  // Call function to build tensor and run model
  run<float, 2, 2>(device, model, input_gpu, shape, result_gpu, result_shape);

  // Compare if values are the same as for python script
  hipMemcpy(result_cpu, result_gpu, sizeof(result_cpu), hipMemcpyDeviceToHost);
  for (int i = 0; i < result_shape[0]; i++) {
    for (int j = 0; j < result_shape[1]; j++) {
      CPPUNIT_ASSERT(std::abs(result_cpu[i][j] - result_check[i][j]) <= 1.0e-05);
    }
  }

}
