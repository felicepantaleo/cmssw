#include <cstdlib>
#include <cuda/std/array>
#include <cuda/std/ranges>
#include <iostream>
#include <chrono>
#include <array>

#include <hip/hip_runtime.h>

#include <torch/torch.h>

using std::chrono::high_resolution_clock;
using std::chrono::duration_cast;
using std::chrono::duration;
using std::chrono::milliseconds;

template <std::size_t N>
torch::Tensor array_to_tensor(torch::Device device, int* arr, const long int* size) {
  long int arr_size[N];
  long int arr_stride[N];
  std::copy(size, size+N, arr_size);
  std::copy(size, size+N, arr_stride);

  std::shift_right(std::begin(arr_stride), std::end(arr_stride), 1);
  arr_stride[0] = 1;
  arr_stride[N-1] *= arr_stride[N-2];

  auto options = torch::TensorOptions().dtype(torch::kInt).device(device).pinned_memory(true);
  torch::Tensor tensor = torch::from_blob(arr, arr_size, arr_stride, options);

  return tensor;
}

template <std::size_t N>
void print_column_major(int* arr, const long int* size) {
  if (N == 2) {
    for (int i = 0; i < size[0]; i++) {
      for (int j = 0; j < size[1]; j++) {
          std::cout << arr[i + j*size[0]] << " ";
      }
      std::cout << std::endl;
    } 
  } else if (N == 3) {
    for (int i = 0; i < size[0]; i++) {
      std::cout << "(" << i << ", .., ..)" << std::endl;
      for (int j = 0; j < size[1]; j++) {
        for (int k = 0; k < size[2]; k++) {
          std::cout << arr[i + j*size[0] + k*size[0]*size[1]] << " ";
        }
        std::cout << std::endl;
      } 
      std::cout << std::endl;
    }
  }
}


int main(int argc, char* argv[]) {  
  torch::Device device(torch::kCUDA);

  int a_cpu[] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15, 16, 17, 18, 19, 20, 21, 22, 23, 24};
  // 2 Dimensional Example
  long int a_shape[] = {4, 6};

  // 3 Dimensional Example
  // const long int a_shape[] = {4, 3, 2};

  const size_t dims = sizeof(a_shape) / sizeof(long int);

  // Prints array in correct form.
  print_column_major<dims>(a_cpu, a_shape);

  int *a_gpu;
  hipMalloc(&a_gpu, sizeof(a_cpu));
  hipMemcpy(a_gpu, a_cpu, sizeof(a_cpu), hipMemcpyHostToDevice);

  // bad behaviour
  auto options = torch::TensorOptions().dtype(torch::kInt).device(device).pinned_memory(true);
  std::cout << "Converting vector to Torch tensors on CPU without stride" << std::endl;
  torch::Tensor tensor = torch::from_blob(a_gpu, a_shape, options);
  std::cout << tensor << std::endl;  

  // Correct Transposition to get to smae dimensions as column major.
  std::cout << "Correct Tensor with Transpose" << std::endl;
  long int a_size[dims];
  std::copy(a_shape, a_shape+dims, a_size);
  std::reverse(std::begin(a_size), std::end(a_size));
  tensor = torch::from_blob(a_gpu, a_size, options);

  tensor = torch::transpose(tensor, 0, dims-1);
  std::cout << tensor << std::endl;

  // Use stride to read correctly.
  std::cout << "Converting vector to Torch tensors on CPU with stride" << std::endl;
  std::cout << array_to_tensor<dims>(device, a_gpu, a_shape) << std::endl;

  long int b_shape[] = {500, 1000};
  int b[b_shape[0]][b_shape[1]];

  for (int i = 0; i < b_shape[0]; i++) {
    for (int j = 0; j < b_shape[1]; j++) {
      b[i][j] = rand();
    }
  } 

  int *b_gpu;
  hipMalloc(&b_gpu, b_shape[0] * b_shape[1] * sizeof(int));
  hipMemcpy(b_gpu, b, b_shape[0] * b_shape[1] * sizeof(int), hipMemcpyHostToDevice);

  std::cout << "Benchmark stride and transpose:" << std::endl;

  auto t1 = high_resolution_clock::now();
  const size_t dim_b = sizeof(b_shape) / sizeof(long int);
  torch::Tensor tensor_stride = array_to_tensor<dim_b>(device, b_gpu, b_shape);
  auto t2 = high_resolution_clock::now();
  duration<double, std::milli> ms_double = t2 - t1;
  std::cout << "Stride:" << ms_double.count() << "ms\n";

  t1 = high_resolution_clock::now();
  const size_t dim_b2 = sizeof(b_shape) / sizeof(long int);
  long int b_size[dim_b2];
  std::copy(b_shape, b_shape+dim_b2, b_size);
  std::reverse(std::begin(a_size), std::end(a_size));

  std::reverse(std::begin(b_shape), std::end(b_shape));
  torch::Tensor tensor_transp = torch::from_blob(b_gpu, b_size, options);
  tensor_transp = torch::transpose(tensor_transp, 0, dim_b2-1);
  t2 = high_resolution_clock::now();
  ms_double = t2 - t1;
  std::cout << "Transpose:" << ms_double.count() << "ms\n";

  return 0;
}
