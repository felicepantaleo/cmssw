//
// Author: Felice Pantaleo, CERN
//

#include "CAHitQuadrupletGeneratorGPU.h"
#include "GPUCACell.h"


void CAHitQuadrupletGeneratorGPU::deallocateOnGPU()
{
  hipStreamDestroy(cudaStream_);

  hipHostFree(h_indices);
  hipHostFree(h_doublets);
  hipHostFree(h_x);
  hipHostFree(h_y);
  hipHostFree(h_z);
  hipHostFree(h_rootLayerPairs);
  hipHostFree(h_foundNtuplets);
  hipHostFree(tmp_layers);
  hipHostFree(tmp_layerDoublets);
  hipHostFree(h_layers);

  hipFree(d_indices);
  hipFree(d_doublets);
  hipFree(d_x);
  hipFree(d_y);
  hipFree(d_z);
  hipFree(d_rootLayerPairs);
  hipFree(device_theCells);
  hipFree(device_isOuterHitOfCell);
  hipFree(d_foundNtuplets);
}

void CAHitQuadrupletGeneratorGPU::allocateOnGPU()
{
  hipStreamCreateWithFlags(&cudaStream_, hipStreamNonBlocking);
  unsigned int maxNumberOfLayerPairs = 13;
  unsigned int maxNumberOfLayers = 10;
  unsigned int maxNumberOfHits = 2000;
  unsigned int maxNumberOfRootLayerPairs = 13;

  hipHostMalloc(&h_doublets, maxNumberOfLayerPairs * sizeof(GPULayerDoublets));

  unsigned int maxNumberOfDoublets = 1;

  hipHostMalloc(&h_indices, maxNumberOfLayerPairs * maxNumberOfDoublets * 2 * sizeof(int));
  hipHostMalloc(&h_x, maxNumberOfLayers * maxNumberOfHits * sizeof(float));
  hipHostMalloc(&h_y, maxNumberOfLayers * maxNumberOfHits * sizeof(float));
  hipHostMalloc(&h_z, maxNumberOfLayers * maxNumberOfHits * sizeof(float));
  hipHostMalloc(&h_rootLayerPairs, maxNumberOfRootLayerPairs * sizeof(int));
  hipHostMalloc(&h_foundNtuplets,
                 sizeof(GPUSimpleVector<maxNumberOfQuadruplets, Quadruplet>));

  hipMalloc(&d_indices,
             maxNumberOfLayerPairs * maxNumberOfDoublets * 2 * sizeof(int));
  hipMalloc(&d_doublets, maxNumberOfLayerPairs * sizeof(GPULayerDoublets));
  hipMalloc(&d_layers, maxNumberOfLayers * sizeof(GPULayerHits));
  hipMalloc(&d_x, maxNumberOfLayers * maxNumberOfHits * sizeof(float));
  hipMalloc(&d_y, maxNumberOfLayers * maxNumberOfHits * sizeof(float));
  hipMalloc(&d_z, maxNumberOfLayers * maxNumberOfHits * sizeof(float));
  hipMalloc(&d_rootLayerPairs,
             maxNumberOfRootLayerPairs * sizeof(unsigned int));
  //////////////////////////////////////////////////////////
  // ALLOCATIONS FOR THE INTERMEDIATE RESULTS (STAYS ON WORKER)
  //////////////////////////////////////////////////////////

  hipMalloc(&device_theCells,
             maxNumberOfLayerPairs * maxNumberOfDoublets * sizeof(GPUCACell));

  hipMalloc(&device_isOuterHitOfCell,
             maxNumberOfLayers * maxNumberOfHits *
                 sizeof(GPUSimpleVector<maxCellsPerHit, unsigned int>));
  hipMemset(device_isOuterHitOfCell, 0,
             maxNumberOfLayers * maxNumberOfHits *
                 sizeof(GPUSimpleVector<maxCellsPerHit, unsigned int>));

  hipMalloc(&d_foundNtuplets,
             sizeof(GPUSimpleVector<maxNumberOfQuadruplets, Quadruplet>));

  hipHostMalloc(&tmp_layers,
                   maxNumberOfLayers * sizeof(GPULayerHits));
  hipHostMalloc(&tmp_layerDoublets,
                   maxNumberOfLayerPairs * sizeof(GPULayerDoublets));
  hipHostMalloc(&h_layers, maxNumberOfLayers * sizeof(GPULayerHits));

}

void CAHitQuadrupletGeneratorGPU::launchKernels()
{



}
