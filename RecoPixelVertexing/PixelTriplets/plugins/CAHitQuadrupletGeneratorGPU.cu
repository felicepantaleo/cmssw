#include "hip/hip_runtime.h"
//
// Author: Felice Pantaleo, CERN
//

#include <cstdint>
#include <hip/hip_runtime.h>
#include <memory>

#include "HeterogeneousCore/CUDAUtilities/interface/cudaCheck.h"
#include "RecoLocalTracker/SiPixelRecHits/interface/pixelCPEforGPU.h"
#include "DataFormats/TrackReco/interface/Track.h"
#include "CAHitQuadrupletGeneratorGPU.h"
#include "GPUCACell.h"
#include "gpuPixelDoublets.h"

using HitsOnCPU = siPixelRecHitsHeterogeneousProduct::HitsOnCPU;
using namespace Eigen;

__global__ void
KernelFastFitAllHits(GPU::SimpleVector<Quadruplet> * foundNtuplets,
    siPixelRecHitsHeterogeneousProduct::HitsOnGPU const * hhp,
    int hits_in_fit,
    float B,
    Rfit::helix_fit *results,
    Rfit::Matrix3xNd *hits,
    Rfit::Matrix3Nd *hits_cov,
    Rfit::circle_fit *circle_fit,
    Vector4d *fast_fit,
    Rfit::line_fit *line_fit)
{
  int helix_start = (blockIdx.x * blockDim.x + threadIdx.x);
  if (helix_start >= foundNtuplets->size()) {
    return;
  }

#ifdef GPU_DEBUG
  printf("BlockDim.x: %d, BlockIdx.x: %d, threadIdx.x: %d, helix_start: %d, cumulative_size: %d\n",
      blockDim.x, blockIdx.x, threadIdx.x, helix_start, foundNtuplets->size());
#endif

  hits[helix_start].resize(3, hits_in_fit);
  hits_cov[helix_start].resize(3 * hits_in_fit, 3 * hits_in_fit);

  // Prepare data structure
  for (unsigned int i = 0; i < hits_in_fit; ++i) {
    auto hit = (*foundNtuplets)[helix_start].hitId[i];
    //  printf("Hit global_x: %f\n", hhp->xg_d[hit]);
    float ge[6];
    hhp->cpeParams->detParams(hhp->detInd_d[hit]).frame.toGlobal(hhp->xerr_d[hit], 0, hhp->yerr_d[hit], ge);
    //  printf("Error: %d: %f,%f,%f,%f,%f,%f\n",hhp->detInd_d[hit],ge[0],ge[1],ge[2],ge[3],ge[4],ge[5]);

    hits[helix_start].col(i) << hhp->xg_d[hit], hhp->yg_d[hit], hhp->zg_d[hit];

    for (auto j = 0; j < 3; ++j) {
      for (auto l = 0; l < 3; ++l) {
        // Index numerology:
        // i: index of the hits/point (0,..,3)
        // j: index of space component (x,y,z)
        // l: index of space components (x,y,z)
        // ge is always in sync with the index i and is formatted as:
        // ge[] ==> [xx, xy, xz, yy, yz, zz]
        // in (j,l) notation, we have:
        // ge[] ==> [(0,0), (0,1), (0,2), (1,1), (1,2), (2,2)]
        // so the index ge_idx corresponds to the matrix elements:
        // | 0  1  2 |
        // | 1  3  4 |
        // | 2  4  5 |
        auto ge_idx = j + l + (j > 0 and l > 0);
        hits_cov[helix_start](i + j * hits_in_fit, i + l * hits_in_fit) = ge[ge_idx];
      }
    }
  }
  fast_fit[helix_start] = Rfit::Fast_fit(hits[helix_start]);
}

__global__ void
KernelCircleFitAllHits(GPU::SimpleVector<Quadruplet> * foundNtuplets,
    int hits_in_fit,
    float B,
    Rfit::helix_fit *results,
    Rfit::Matrix3xNd *hits,
    Rfit::Matrix3Nd *hits_cov,
    Rfit::circle_fit *circle_fit,
    Vector4d *fast_fit,
    Rfit::line_fit *line_fit)
{
  int helix_start = (blockIdx.x * blockDim.x + threadIdx.x);
  if (helix_start >= foundNtuplets->size()) {
    return;
  }

#ifdef GPU_DEBUG
    printf("BlockDim.x: %d, BlockIdx.x: %d, threadIdx.x: %d, helix_start: %d"
           "cumulative_size: %d\n",
           blockDim.x, blockIdx.x, threadIdx.x, helix_start, foundNtuplets->size());
#endif
  u_int n = hits[helix_start].cols();

  Rfit::VectorNd rad = (hits[helix_start].block(0, 0, 2, n).colwise().norm());

  circle_fit[helix_start] =
      Rfit::Circle_fit(hits[helix_start].block(0, 0, 2, n),
                       hits_cov[helix_start].block(0, 0, 2 * n, 2 * n),
                       fast_fit[helix_start], rad, B, true);

#ifdef GPU_DEBUG
    printf("KernelCircleFitAllHits circle.par(0): %d %f\n", helix_start,
           circle_fit[helix_start].par(0));
    printf("KernelCircleFitAllHits circle.par(1): %d %f\n", helix_start,
           circle_fit[helix_start].par(1));
    printf("KernelCircleFitAllHits circle.par(2): %d %f\n", helix_start,
           circle_fit[helix_start].par(2));
#endif
}

__global__ void
KernelLineFitAllHits(GPU::SimpleVector<Quadruplet> * foundNtuplets,
    float B,
    Rfit::helix_fit *results,
    Rfit::Matrix3xNd *hits,
    Rfit::Matrix3Nd *hits_cov,
    Rfit::circle_fit *circle_fit,
    Vector4d *fast_fit,
    Rfit::line_fit *line_fit)
{
  int helix_start = (blockIdx.x * blockDim.x + threadIdx.x);
  if (helix_start >= foundNtuplets->size()) {
    return;
  }

#ifdef GPU_DEBUG

    printf("BlockDim.x: %d, BlockIdx.x: %d, threadIdx.x: %d, helix_start: %d, "
           "cumulative_size: %d\n",
           blockDim.x, blockIdx.x, threadIdx.x, helix_start, foundNtuplets->size());
#endif

  line_fit[helix_start] =
      Rfit::Line_fit(hits[helix_start], hits_cov[helix_start],
                     circle_fit[helix_start], fast_fit[helix_start], B, true);

  par_uvrtopak(circle_fit[helix_start], B, true);

  // Grab helix_fit from the proper location in the output vector
  Rfit::helix_fit &helix = results[helix_start];
  helix.par << circle_fit[helix_start].par, line_fit[helix_start].par;

  // TODO: pass properly error booleans

  helix.cov = MatrixXd::Zero(5, 5);
  helix.cov.block(0, 0, 3, 3) = circle_fit[helix_start].cov;
  helix.cov.block(3, 3, 2, 2) = line_fit[helix_start].cov;

  helix.q = circle_fit[helix_start].q;
  helix.chi2_circle = circle_fit[helix_start].chi2;
  helix.chi2_line = line_fit[helix_start].chi2;

#ifdef GPU_DEBUG

    printf("KernelLineFitAllHits line.par(0): %d %f\n", helix_start,
           circle_fit[helix_start].par(0));
    printf("KernelLineFitAllHits line.par(1): %d %f\n", helix_start,
           line_fit[helix_start].par(1));
#endif
}

// __global__
// void kernel_filterGPUPixelTracks(Rfit::helix_fit * const __restrict__  helix_fit_results,
//   GPU::SimpleVector<Quadruplet> * const __restrict__ foundNtuplets,
//   CAHitNtupletHeterogeneousProduct::GPUProduct* filteredPixelTracks, float fitMaxChi2, float fitMaxTip, float nSigmaTipMaxTolerance )
// {
//   const auto nSeeds = foundNtuplets->size();
//   auto idx = threadIdx.x + blockIdx.x * blockDim.x;
//   if(idx < nSeeds)
//   {
//     const auto& helix = helix_fit_results[idx];
//     float valTip = helix.par(1);
//     float errTip = std::sqrt(helix.cov(1, 1));
//     if( (std::abs(valTip) - fitMaxTip < nSigmaTipMaxTolerance * errTip ) and
//         (helix.chi2_circle + helix.chi2_line <= fitMaxChi2))
//     {
//       filteredPixelTracks->emplace_back(CAHitNtupletHeterogeneousProduct::GPUPixelTrack{helix, (*foundNtuplets)[idx]});
//     }
//   }
// }


__global__
void kernel_makeGPUPixelTracks(Rfit::helix_fit * const __restrict__  helix_fit_results,
  Rfit::circle_fit * const __restrict__ circle_fit_results,
  GPU::SimpleVector<Quadruplet> * const __restrict__ foundNtuplets,
  FakeRecoTrack* recoTracks)
{
  const auto nSeeds = foundNtuplets->size();
  auto idx = threadIdx.x + blockIdx.x * blockDim.x;
  if(idx < nSeeds)
  {
    const auto& helix = helix_fit_results[idx];
    auto& recoTrack = recoTracks[idx];
    recoTrack.charge = circle_fit_results[idx].q;
    recoTrack.chi2 = helix.chi2_circle + helix.chi2_line;
    recoTrack.ndof = 3;
    recoTrack.covariance[0] =  helix.cov(0,0);
    recoTrack.covariance[1] =  helix.cov(0,1);
    recoTrack.covariance[2] =  helix.cov(1,1);
    recoTrack.covariance[3] =  helix.cov(0,2);
    recoTrack.covariance[4] =  helix.cov(1,2);
    recoTrack.covariance[5] =  helix.cov(2,2);
    recoTrack.covariance[6] =  helix.cov(0,3);
    recoTrack.covariance[7] =  helix.cov(1,3);
    recoTrack.covariance[8] =  helix.cov(2,3);
    recoTrack.covariance[9] =  helix.cov(3,3);
    recoTrack.covariance[10] = helix.cov(0,4);
    recoTrack.covariance[11] = helix.cov(1,4);
    recoTrack.covariance[12] = helix.cov(2,4);
    recoTrack.covariance[13] = helix.cov(3,4);
    recoTrack.covariance[14] = helix.cov(4,4);
    float valTip = helix.par(1);
    float errTip = std::sqrt(helix.cov(1, 1));

    //TODO: Fill the parameters of the track and point of closest approach position
    // recoTrack.vertexPos[0] =
    // Vector5d par;  //!<(phi,Tip,pt,cotan(theta)),Zip)
    // Matrix5d cov;
    // /*!< ()->cov() \n
    //   |(phi,phi)|(Tip,phi)|(p_t,phi)|(c_t,phi)|(Zip,phi)| \n
    //   |(phi,Tip)|(Tip,Tip)|(p_t,Tip)|(c_t,Tip)|(Zip,Tip)| \n
    //   |(phi,p_t)|(Tip,p_t)|(p_t,p_t)|(c_t,p_t)|(Zip,p_t)| \n
    //   |(phi,c_t)|(Tip,c_t)|(p_t,c_t)|(c_t,c_t)|(Zip,c_t)| \n
    //   |(phi,Zip)|(Tip,Zip)|(p_t,Zip)|(c_t,Zip)|(Zip,Zip)|
    // */
  }
}


__global__ void
kernel_checkOverflows(GPU::SimpleVector<Quadruplet> *foundNtuplets,
               GPUCACell *cells, uint32_t const * nCells,
               GPU::VecArray< unsigned int, 256> *isOuterHitOfCell,
               uint32_t nHits, uint32_t maxNumberOfDoublets) {

 auto idx = threadIdx.x + blockIdx.x * blockDim.x;
 #ifdef GPU_DEBUG
 if (0==idx)
   printf("number of found cells %d\n",*nCells);
 #endif
 if (idx < (*nCells) ) {
   auto &thisCell = cells[idx];
   if (thisCell.theOuterNeighbors.full()) //++tooManyNeighbors[thisCell.theLayerPairId];
     printf("OuterNeighbors overflow %d in %d\n", idx, thisCell.theLayerPairId);
 }
 if (idx < nHits) {
   if (isOuterHitOfCell[idx].full()) // ++tooManyOuterHitOfCell;
     printf("OuterHitOfCell overflow %d\n", idx);
 }
}


__global__ void
kernel_connect(GPU::SimpleVector<Quadruplet> *foundNtuplets,
               GPUCACell *cells, uint32_t const * nCells,
               GPU::VecArray< unsigned int, 256> *isOuterHitOfCell,
               float ptmin,
               float region_origin_radius, const float thetaCut,
               const float phiCut, const float hardPtCut,
               unsigned int maxNumberOfDoublets, unsigned int maxNumberOfHits_) {

  float region_origin_x = 0.;
  float region_origin_y = 0.;

  auto cellIndex = threadIdx.x + blockIdx.x * blockDim.x;

  if (0==cellIndex) foundNtuplets->reset(); // ready for next kernel

  if (cellIndex >= (*nCells) ) return;
  auto &thisCell = cells[cellIndex];
  auto innerHitId = thisCell.get_inner_hit_id();
  auto numberOfPossibleNeighbors = isOuterHitOfCell[innerHitId].size();
  for (auto j = 0; j < numberOfPossibleNeighbors; ++j) {
     auto otherCell = isOuterHitOfCell[innerHitId][j];

     if (thisCell.check_alignment_and_tag(
                 cells, otherCell, ptmin, region_origin_x, region_origin_y,
                  region_origin_radius, thetaCut, phiCut, hardPtCut)
        ) {
          cells[otherCell].theOuterNeighbors.push_back(cellIndex);
     }
  }
}

__global__ void kernel_find_ntuplets(
    GPUCACell *cells, uint32_t const * nCells,
    GPU::SimpleVector<Quadruplet> *foundNtuplets,
    unsigned int minHitsPerNtuplet,
    unsigned int maxNumberOfDoublets)
{

  auto cellIndex = threadIdx.x + blockIdx.x * blockDim.x;
  if (cellIndex >= (*nCells)) return;
  auto &thisCell = cells[cellIndex];
  if (thisCell.theLayerPairId!=0 && thisCell.theLayerPairId!=3 && thisCell.theLayerPairId!=8) return; // inner layer is 0 FIXME
  GPU::VecArray<unsigned int, 3> stack;
  stack.reset();
  thisCell.find_ntuplets(cells, foundNtuplets, stack, minHitsPerNtuplet);
  assert(stack.size()==0);
  // printf("in %d found quadruplets: %d\n", cellIndex, foundNtuplets->size());
}

__global__ void
kernel_print_found_ntuplets(GPU::SimpleVector<Quadruplet> *foundNtuplets, int maxPrint) {
  for (int i = 0; i < std::min(maxPrint, foundNtuplets->size()); ++i) {
    printf("\nquadruplet %d: %d %d %d %d\n", i,
           (*foundNtuplets)[i].hitId[0],
           (*foundNtuplets)[i].hitId[1],
           (*foundNtuplets)[i].hitId[2],
           (*foundNtuplets)[i].hitId[3]
          );
  }
}

void CAHitQuadrupletGeneratorGPU::deallocateOnGPU()
{
  for (size_t i = 0; i < h_foundNtupletsVec_.size(); ++i)
  {
    hipHostFree(h_foundNtupletsVec_[i]);
    hipHostFree(h_foundNtupletsData_[i]);
    hipFree(d_foundNtupletsVec_[i]);
    hipFree(d_foundNtupletsData_[i]);
    hipHostFree(h_foundGPUPixelTracks_[i]);
    hipFree(d_foundGPUPixelTracks_[i]);
  }

  hipFree(device_theCells_);
  hipFree(device_isOuterHitOfCell_);
  hipFree(device_nCells_);

  // Free Riemann Fit stuff
  hipFree(hitsGPU_);
  hipFree(hits_covGPU_);
  hipFree(fast_fit_resultsGPU_);
  hipFree(circle_fit_resultsGPU_);
  hipFree(line_fit_resultsGPU_);
  hipFree(helix_fit_resultsGPU_);
  hipFree(d_recoTracks_);
}



void CAHitQuadrupletGeneratorGPU::allocateOnGPU()
{
  //////////////////////////////////////////////////////////
  // ALLOCATIONS FOR THE INTERMEDIATE RESULTS (STAYS ON WORKER)
  //////////////////////////////////////////////////////////

  cudaCheck(hipMalloc(&device_theCells_,
             PixelGPUConstants::MaxNumOfDoublets * sizeof(GPUCACell)));
  cudaCheck(hipMalloc(&device_nCells_, sizeof(uint32_t)));
  cudaCheck(hipMemset(device_nCells_, 0, sizeof(uint32_t)));

  cudaCheck(hipMalloc(&device_isOuterHitOfCell_,
             PixelGPUConstants::maxNumberOfHits * sizeof(GPU::VecArray<unsigned int, maxCellsPerHit_>)));
  cudaCheck(hipMemset(device_isOuterHitOfCell_, 0,
             PixelGPUConstants::maxNumberOfHits * sizeof(GPU::VecArray<unsigned int, maxCellsPerHit_>)));

  h_foundNtupletsVec_.resize(maxNumberOfRegions_);
  h_foundNtupletsData_.resize(maxNumberOfRegions_);
  d_foundNtupletsVec_.resize(maxNumberOfRegions_);
  d_foundNtupletsData_.resize(maxNumberOfRegions_);
  h_foundGPUPixelTracks_.resize(maxNumberOfRegions_);
  d_foundGPUPixelTracks_.resize(maxNumberOfRegions_);

  // FIXME this could be rewritten with a single pair of hipHostMalloc / hipMalloc
  for (int i = 0; i < maxNumberOfRegions_; ++i) {
    cudaCheck(hipHostMalloc(&h_foundNtupletsData_[i],  sizeof(Quadruplet) * PixelGPUConstants::maxNumberOfQuadruplets));
    cudaCheck(hipHostMalloc(&h_foundNtupletsVec_[i],   sizeof(GPU::SimpleVector<Quadruplet>)));
    new(h_foundNtupletsVec_[i]) GPU::SimpleVector<Quadruplet>(PixelGPUConstants::maxNumberOfQuadruplets, h_foundNtupletsData_[i]);
    cudaCheck(hipMalloc(&d_foundNtupletsData_[i],      sizeof(Quadruplet) * PixelGPUConstants::maxNumberOfQuadruplets));
    cudaCheck(hipMemset(d_foundNtupletsData_[i], 0x00, sizeof(Quadruplet) * PixelGPUConstants::maxNumberOfQuadruplets));
    cudaCheck(hipMalloc(&d_foundNtupletsVec_[i],       sizeof(GPU::SimpleVector<Quadruplet>)));
    GPU::SimpleVector<Quadruplet> tmp_foundNtuplets(PixelGPUConstants::maxNumberOfQuadruplets, d_foundNtupletsData_[i]);
    cudaCheck(hipMemcpy(d_foundNtupletsVec_[i], & tmp_foundNtuplets, sizeof(GPU::SimpleVector<Quadruplet>), hipMemcpyDefault));

    cudaCheck(hipHostMalloc(&h_foundGPUPixelTracks_[i],sizeof(CAHitNtupletHeterogeneousProduct::GPUProduct)));
    cudaCheck(hipMalloc(&d_foundGPUPixelTracks_[i],sizeof(CAHitNtupletHeterogeneousProduct::GPUProduct)));
    cudaCheck(hipMemset(d_foundGPUPixelTracks_[i], 0x00, sizeof(CAHitNtupletHeterogeneousProduct::GPUProduct)));

  }

  // Riemann-Fit related allocations
  cudaCheck(hipMalloc(&hitsGPU_, 48 * PixelGPUConstants::maxNumberOfQuadruplets * sizeof(Rfit::Matrix3xNd(3, 4))));
  cudaCheck(hipMemset(hitsGPU_, 0x00, 48 * PixelGPUConstants::maxNumberOfQuadruplets * sizeof(Rfit::Matrix3xNd(3, 4))));

  cudaCheck(hipMalloc(&hits_covGPU_, 48 * PixelGPUConstants::maxNumberOfQuadruplets * sizeof(Rfit::Matrix3Nd(12, 12))));
  cudaCheck(hipMemset(hits_covGPU_, 0x00, 48 * PixelGPUConstants::maxNumberOfQuadruplets * sizeof(Rfit::Matrix3Nd(12, 12))));

  cudaCheck(hipMalloc(&fast_fit_resultsGPU_, 48 * PixelGPUConstants::maxNumberOfQuadruplets * sizeof(Vector4d)));
  cudaCheck(hipMemset(fast_fit_resultsGPU_, 0x00, 48 * PixelGPUConstants::maxNumberOfQuadruplets * sizeof(Vector4d)));

  cudaCheck(hipMalloc(&circle_fit_resultsGPU_, 48 * PixelGPUConstants::maxNumberOfQuadruplets * sizeof(Rfit::circle_fit)));
  cudaCheck(hipMemset(circle_fit_resultsGPU_, 0x00, 48 * PixelGPUConstants::maxNumberOfQuadruplets * sizeof(Rfit::circle_fit)));

  cudaCheck(hipMalloc(&line_fit_resultsGPU_, PixelGPUConstants::maxNumberOfQuadruplets * sizeof(Rfit::line_fit)));
  cudaCheck(hipMemset(line_fit_resultsGPU_, 0x00, PixelGPUConstants::maxNumberOfQuadruplets * sizeof(Rfit::line_fit)));

  cudaCheck(hipMalloc(&helix_fit_resultsGPU_, sizeof(Rfit::helix_fit)*PixelGPUConstants::maxNumberOfQuadruplets));
  cudaCheck(hipMemset(helix_fit_resultsGPU_, 0x00, sizeof(Rfit::helix_fit)*PixelGPUConstants::maxNumberOfQuadruplets));

  cudaCheck(hipMalloc(&d_recoTracks_, sizeof(FakeRecoTrack)*PixelGPUConstants::maxNumberOfQuadruplets)));
  h_product_.fitResults = helix_fit_resultsGPU_;
  h_product_.d_foundNtuplets = d_foundNtupletsVec_;
  h_product.d_foundNtupletsData = d_foundNtupletsData_;
  h_product.d_recoTracks = d_recoTracks_;

}

void CAHitQuadrupletGeneratorGPU::launchKernels(const TrackingRegion &region,
                                                int regionIndex, HitsOnCPU const & hh,
                                                bool transferToCPU,
                                                hipStream_t cudaStream)
{
  assert(regionIndex < maxNumberOfRegions_);
  assert(0==regionIndex);

  h_foundNtupletsVec_[regionIndex]->reset();

  auto nhits = hh.nHits;
  assert(nhits <= PixelGPUConstants::maxNumberOfHits);
  auto numberOfBlocks = (PixelGPUConstants::MaxNumOfDoublets + 512 - 1)/512;
  kernel_connect<<<numberOfBlocks, 512, 0, cudaStream>>>(
      d_foundNtupletsVec_[regionIndex], // needed only to be reset, ready for next kernel
      device_theCells_, device_nCells_,
      device_isOuterHitOfCell_,
      region.ptMin(),
      region.originRBound(), caThetaCut, caPhiCut, caHardPtCut,
      PixelGPUConstants::MaxNumOfDoublets, PixelGPUConstants::maxNumberOfHits
  );
  cudaCheck(hipGetLastError());

  kernel_find_ntuplets<<<numberOfBlocks, 512, 0, cudaStream>>>(
      device_theCells_, device_nCells_,
      d_foundNtupletsVec_[regionIndex],
      4, PixelGPUConstants::MaxNumOfDoublets);
  cudaCheck(hipGetLastError());


  numberOfBlocks = (std::max(int(nhits), PixelGPUConstants::MaxNumOfDoublets) + 512 - 1)/512;
  kernel_checkOverflows<<<numberOfBlocks, 512, 0, cudaStream>>>(
                        d_foundNtupletsVec_[regionIndex],
                        device_theCells_, device_nCells_,
                        device_isOuterHitOfCell_, nhits,
                        PixelGPUConstants::MaxNumOfDoublets
                       );


  // kernel_print_found_ntuplets<<<1, 1, 0, cudaStream>>>(d_foundNtupletsVec_[regionIndex], 10);


  numberOfBlocks = (PixelGPUConstants::maxNumberOfQuadruplets + 512 - 1)/512;

  KernelFastFitAllHits<<<numberOfBlocks, 512, 0, cudaStream>>>(
      d_foundNtupletsVec_[regionIndex], hh.gpu_d, 4, bField_, helix_fit_resultsGPU_,
      hitsGPU_, hits_covGPU_, circle_fit_resultsGPU_, fast_fit_resultsGPU_,
      line_fit_resultsGPU_);
  cudaCheck(hipGetLastError());

  KernelCircleFitAllHits<<<numberOfBlocks, 256, 0, cudaStream>>>(
      d_foundNtupletsVec_[regionIndex], 4, bField_, helix_fit_resultsGPU_,
      hitsGPU_, hits_covGPU_, circle_fit_resultsGPU_, fast_fit_resultsGPU_,
      line_fit_resultsGPU_);
  cudaCheck(hipGetLastError());

  KernelLineFitAllHits<<<numberOfBlocks, 256, 0, cudaStream>>>(
      d_foundNtupletsVec_[regionIndex], bField_, helix_fit_resultsGPU_,
      hitsGPU_, hits_covGPU_, circle_fit_resultsGPU_, fast_fit_resultsGPU_,
      line_fit_resultsGPU_);
  cudaCheck(hipGetLastError());


  kernel_filterGPUPixelTracks<<<numberOfBlocks, 256, 0, cudaStream>>>(
    helix_fit_resultsGPU_, d_foundNtupletsVec_[regionIndex], d_foundGPUPixelTracks_[regionIndex],
    fitMaxChi2_,  fitMaxTip_, nSigmaTipMaxTolerance_ );

  if(transferToCPU) {
    cudaCheck(hipMemcpyAsync(h_foundNtupletsVec_[regionIndex], d_foundNtupletsVec_[regionIndex],
                              sizeof(GPU::SimpleVector<Quadruplet>),
                              hipMemcpyDeviceToHost, cudaStream));

    cudaCheck(hipMemcpyAsync(h_foundNtupletsData_[regionIndex], d_foundNtupletsData_[regionIndex],
                              PixelGPUConstants::maxNumberOfQuadruplets*sizeof(Quadruplet),
                              hipMemcpyDeviceToHost, cudaStream));
  }
}

void CAHitQuadrupletGeneratorGPU::cleanup(hipStream_t cudaStream) {
  // this lazily resets temporary memory for the next event, and is not needed for reading the output
  cudaCheck(hipMemsetAsync(device_isOuterHitOfCell_, 0,
                            PixelGPUConstants::maxNumberOfHits * sizeof(GPU::VecArray<unsigned int, maxCellsPerHit_>),
                            cudaStream));
  cudaCheck(hipMemsetAsync(device_nCells_, 0, sizeof(uint32_t), cudaStream));
}

std::vector<std::array<int, 4>>
CAHitQuadrupletGeneratorGPU::fetchKernelResult(int regionIndex)
{
  assert(0==regionIndex);
  h_foundNtupletsVec_[regionIndex]->set_data(h_foundNtupletsData_[regionIndex]);

  std::vector<std::array<int, 4>> quadsInterface(h_foundNtupletsVec_[regionIndex]->size());
  for (int i = 0; i < h_foundNtupletsVec_[regionIndex]->size(); ++i) {
    for (int j = 0; j<4; ++j) quadsInterface[i][j] = (*h_foundNtupletsVec_[regionIndex])[i].hitId[j];
  }
  return quadsInterface;
}

void CAHitQuadrupletGeneratorGPU::buildDoublets(HitsOnCPU const & hh, hipStream_t stream) {
  auto nhits = hh.nHits;

  int threadsPerBlock = gpuPixelDoublets::getDoubletsFromHistoMaxBlockSize;
  int blocks = (3 * nhits + threadsPerBlock - 1) / threadsPerBlock;
  gpuPixelDoublets::getDoubletsFromHisto<<<blocks, threadsPerBlock, 0, stream>>>(device_theCells_, device_nCells_, hh.gpu_d, device_isOuterHitOfCell_);
  cudaCheck(hipGetLastError());
}
