#include "hip/hip_runtime.h"
//
// Author: Felice Pantaleo, CERN
//

#include "CAHitQuadrupletGeneratorGPU.h"
#include "GPUCACell.h"


template<int maxNumberOfQuadruplets>
__global__
void kernel_create(const unsigned int numberOfLayerPairs, const GPULayerDoublets* gpuDoublets,
        const GPULayerHits* gpuHitsOnLayers, GPUCACell* cells, GPUSimpleVector<200, unsigned int> * isOuterHitOfCell,
        GPUSimpleVector<maxNumberOfQuadruplets, Quadruplet>* foundNtuplets, const float region_origin_x,const float region_origin_y,
        unsigned int maxNumberOfDoublets, unsigned int maxNumberOfHits)
{

    unsigned int layerPairIndex = blockIdx.y;
    unsigned int cellIndexInLayerPair = threadIdx.x + blockIdx.x * blockDim.x;
    if(cellIndexInLayerPair == 0 && layerPairIndex == 0)
    {
        foundNtuplets->reset();
    }

    if (layerPairIndex < numberOfLayerPairs)
    {
        int outerLayerId = gpuDoublets[layerPairIndex].outerLayerId;
        auto globalFirstDoubletIdx = layerPairIndex*maxNumberOfDoublets;
        auto globalFirstHitIdx = outerLayerId*maxNumberOfHits;

        for (unsigned int i = cellIndexInLayerPair; i < gpuDoublets[layerPairIndex].size;
                i += gridDim.x * blockDim.x)
        {
            auto globalCellIdx = i+globalFirstDoubletIdx;
            auto& thisCell = cells[globalCellIdx];
            auto outerHitId = gpuDoublets[layerPairIndex].indices[2 * i + 1];
            thisCell.init(&gpuDoublets[layerPairIndex], gpuHitsOnLayers, layerPairIndex, globalCellIdx,
                    gpuDoublets[layerPairIndex].indices[2 * i], outerHitId, region_origin_x,  region_origin_y);

            isOuterHitOfCell[globalFirstHitIdx+outerHitId].push_back_ts(globalCellIdx);
        }
    }
}




void CAHitQuadrupletGeneratorGPU::deallocateOnGPU()
{
  hipStreamDestroy(cudaStream_);

  hipHostFree(h_indices);
  hipHostFree(h_doublets);
  hipHostFree(h_x);
  hipHostFree(h_y);
  hipHostFree(h_z);
  hipHostFree(h_rootLayerPairs);
  hipHostFree(h_foundNtuplets);
  hipHostFree(tmp_layers);
  hipHostFree(tmp_layerDoublets);
  hipHostFree(h_layers);

  hipFree(d_indices);
  hipFree(d_doublets);
  hipFree(d_x);
  hipFree(d_y);
  hipFree(d_z);
  hipFree(d_rootLayerPairs);
  hipFree(device_theCells);
  hipFree(device_isOuterHitOfCell);
  hipFree(d_foundNtuplets);
}

void CAHitQuadrupletGeneratorGPU::allocateOnGPU()
{
  hipStreamCreateWithFlags(&cudaStream_, hipStreamNonBlocking);
  unsigned int maxNumberOfLayerPairs = 13;
  unsigned int maxNumberOfLayers = 10;
  unsigned int maxNumberOfHits = 2000;
  unsigned int maxNumberOfRootLayerPairs = 13;

  hipHostMalloc(&h_doublets, maxNumberOfLayerPairs * sizeof(GPULayerDoublets));

  unsigned int maxNumberOfDoublets = 1;

  hipHostMalloc(&h_indices, maxNumberOfLayerPairs * maxNumberOfDoublets * 2 * sizeof(int));
  hipHostMalloc(&h_x, maxNumberOfLayers * maxNumberOfHits * sizeof(float));
  hipHostMalloc(&h_y, maxNumberOfLayers * maxNumberOfHits * sizeof(float));
  hipHostMalloc(&h_z, maxNumberOfLayers * maxNumberOfHits * sizeof(float));
  hipHostMalloc(&h_rootLayerPairs, maxNumberOfRootLayerPairs * sizeof(int));
  hipHostMalloc(&h_foundNtuplets,
                 sizeof(GPUSimpleVector<maxNumberOfQuadruplets, Quadruplet>));

  hipMalloc(&d_indices,
             maxNumberOfLayerPairs * maxNumberOfDoublets * 2 * sizeof(int));
  hipMalloc(&d_doublets, maxNumberOfLayerPairs * sizeof(GPULayerDoublets));
  hipMalloc(&d_layers, maxNumberOfLayers * sizeof(GPULayerHits));
  hipMalloc(&d_x, maxNumberOfLayers * maxNumberOfHits * sizeof(float));
  hipMalloc(&d_y, maxNumberOfLayers * maxNumberOfHits * sizeof(float));
  hipMalloc(&d_z, maxNumberOfLayers * maxNumberOfHits * sizeof(float));
  hipMalloc(&d_rootLayerPairs,
             maxNumberOfRootLayerPairs * sizeof(unsigned int));
  //////////////////////////////////////////////////////////
  // ALLOCATIONS FOR THE INTERMEDIATE RESULTS (STAYS ON WORKER)
  //////////////////////////////////////////////////////////

  hipMalloc(&device_theCells,
             maxNumberOfLayerPairs * maxNumberOfDoublets * sizeof(GPUCACell));

  hipMalloc(&device_isOuterHitOfCell,
             maxNumberOfLayers * maxNumberOfHits *
                 sizeof(GPUSimpleVector<maxCellsPerHit, unsigned int>));
  hipMemset(device_isOuterHitOfCell, 0,
             maxNumberOfLayers * maxNumberOfHits *
                 sizeof(GPUSimpleVector<maxCellsPerHit, unsigned int>));

  hipMalloc(&d_foundNtuplets,
             sizeof(GPUSimpleVector<maxNumberOfQuadruplets, Quadruplet>));

  hipHostMalloc(&tmp_layers,
                   maxNumberOfLayers * sizeof(GPULayerHits));
  hipHostMalloc(&tmp_layerDoublets,
                   maxNumberOfLayerPairs * sizeof(GPULayerDoublets));
  hipHostMalloc(&h_layers, maxNumberOfLayers * sizeof(GPULayerHits));

}

void CAHitQuadrupletGeneratorGPU::launchKernels(const TrackingRegion &region)
{
  dim3 numberOfBlocks_create(32, numberOfLayerPairs);
  dim3 numberOfBlocks_connect(16, numberOfLayerPairs);
  dim3 numberOfBlocks_find(8, numberOfRootLayerPairs);

  kernel_create<<<numberOfBlocks_create,32,0,cudaStream_>>>(numberOfLayerPairs, d_doublets,
                          d_layers, device_theCells,
                          device_isOuterHitOfCell, d_foundNtuplets,region.origin().x(), region.origin().y(), maxNumberOfDoublets, maxNumberOfHits);
}
