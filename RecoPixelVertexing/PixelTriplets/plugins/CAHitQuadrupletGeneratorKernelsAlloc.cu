#include "CAHitQuadrupletGeneratorKernels.h"


void
CAHitQuadrupletGeneratorKernels::deallocateOnGPU()
{

  // crash on multi-gpu processes
  //printCounters();

  hipFree(counters_);

  hipFree(device_theCells_);
  hipFree(device_isOuterHitOfCell_);
  hipFree(device_nCells_);
//  hipFree(device_hitToTuple_);
  hipFree(device_hitToTuple_apc_);
  hipFree(device_tupleMultiplicity_);
  hipFree(device_tmws_);
}

void CAHitQuadrupletGeneratorKernels::allocateOnGPU()
{
  //////////////////////////////////////////////////////////
  // ALLOCATIONS FOR THE INTERMEDIATE RESULTS (STAYS ON WORKER)
  //////////////////////////////////////////////////////////

  cudaCheck(hipMalloc(&counters_, sizeof(Counters)));
  cudaCheck(hipMemset(counters_,0,sizeof(Counters)));

  cudaCheck(hipMalloc(&device_theCells_,
             CAConstants::maxNumberOfLayerPairs() * CAConstants::maxNumberOfDoublets() * sizeof(GPUCACell)));
  cudaCheck(hipMalloc(&device_nCells_, sizeof(uint32_t)));
  cudaCheck(hipMemset(device_nCells_, 0, sizeof(uint32_t)));

  cudaCheck(hipMalloc(&device_isOuterHitOfCell_,
             PixelGPUConstants::maxNumberOfHits * sizeof(CAConstants::OuterHitOfCell)));
  cudaCheck(hipMemset(device_isOuterHitOfCell_, 0,
             PixelGPUConstants::maxNumberOfHits * sizeof(CAConstants::OuterHitOfCell)));

//   cudaCheck(hipMalloc(&device_hitToTuple_, sizeof(HitToTuple)));
   cudaCheck(hipMalloc(&device_hitToTuple_apc_, sizeof(AtomicPairCounter)));

   cudaCheck(hipMalloc(&device_tupleMultiplicity_,sizeof(TupleMultiplicity)));
   cudaCheck(hipMemset(device_tupleMultiplicity_,0,sizeof(TupleMultiplicity))); // overkill
   cudaCheck(hipMalloc(&device_tmws_, TupleMultiplicity::wsSize()));
}

void CAHitQuadrupletGeneratorKernels::cleanup(hipStream_t cudaStream) {
  // this lazily resets temporary memory for the next event, and is not needed for reading the output
  cudaCheck(hipMemsetAsync(device_isOuterHitOfCell_, 0,
                            PixelGPUConstants::maxNumberOfHits * sizeof(CAConstants::OuterHitOfCell),
                            cudaStream));
  cudaCheck(hipMemsetAsync(device_nCells_, 0, sizeof(uint32_t), cudaStream));

  cudautils::launchZero(device_tupleMultiplicity_,cudaStream);

}

